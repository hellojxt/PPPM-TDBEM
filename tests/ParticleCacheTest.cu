#include "hip/hip_runtime.h"
#include <catch2/catch_approx.hpp>
#include <catch2/catch_test_macros.hpp>
#include <catch2/generators/catch_generators.hpp>
#include <vector>
#include "array_writer.h"
#include "bem.h"
#include "case_generator.h"
#include "gui.h"
#include "macro.h"
#include "pppm.h"
#include "sound_source.h"
#include "window.h"

using Catch::Approx;

__global__ void set_signal_kernel(PPPMSolver pppm, SineSource sine)
{
    int particle_idx = blockIdx.x * blockDim.x + threadIdx.x;
    float neumann_amp = 1e10;
    float dirichlet_amp = 1e10;
    float dt = pppm.fdtd.dt;
    float t = pppm.fdtd.t;
    pppm.particle_history[particle_idx].neumann[t] = neumann_amp * sine(dt * t, (particle_idx + 1)).real();
    pppm.particle_history[particle_idx].dirichlet[t] = dirichlet_amp * sine(dt * t, (particle_idx + 1)).imag();
}

TEST_CASE("ParticleCache", "[pc]")
{
    using namespace pppm;
    PPPMSolver *solver = random_pppm(1024, 32);
    solver->precompute_particle_cache();
    auto particle_map = solver->cache.particle_map.cpu();
    auto particle_data = solver->cache.particle_data.cpu();
    auto particles = solver->pg.particles.cpu();
    auto vertices = solver->pg.vertices.cpu();
    REQUIRE(particle_map.size() == particles.size());

    for (int i = 0; i < particles.size(); i++)
    {
        auto particle = particles[i];
        auto r = particle_map[i].range;
        auto base_coord = particle_map[i].base_coord;
        auto dcoord = particle.pos - solver->pg.getCenter(base_coord);
        SECTION("test particle cache info (cache size and neighbor list)")
        {

            REQUIRE((dcoord.x >= 0 && dcoord.x <= solver->fdtd.dl));
            REQUIRE((dcoord.y >= 0 && dcoord.y <= solver->fdtd.dl));
            REQUIRE((dcoord.z >= 0 && dcoord.z <= solver->fdtd.dl));
            float3 center = (solver->pg.getCenter(base_coord) + solver->pg.getCenter(base_coord + 1)) / 2;
            int neighbor_number = 0;
            for (int j = 0; j < particles.size(); j++)
            {
                auto other_particle = particles[j];
                auto other_dcoord = other_particle.pos - center;
                auto max_dim_length = std::max(std::max(abs(other_dcoord.x), abs(other_dcoord.y)), abs(other_dcoord.z));
                if (max_dim_length < solver->fdtd.dl * 2)
                {
                    int same_particle_id_num = 0;
                    for (int k = r.start; k < r.end; k++)
                    {
                        if (particle_data[k].particle_id == j)
                        {
                            same_particle_id_num++;
                        }
                    }
                    REQUIRE(same_particle_id_num == 1);
                    neighbor_number++;
                }
            }
            REQUIRE(r.end - r.start == neighbor_number);
        }
        SECTION("test weights in particle cahe")
        {
            float func_params[8];
            for (int j = 0; j < 8; j++)
                func_params[j] = RAND_F;
            auto func = [func_params](float3 coord) {
                int x = coord.x;
                int y = coord.y;
                int z = coord.z;
                return func_params[0] * x + func_params[1] * y + func_params[2] * z + func_params[3] * x * y +
                       func_params[4] * x * z + func_params[5] * y * z + func_params[6] * x * y * z + func_params[7];
            };
            auto &weights = particle_map[i].weight;
            float interpolation_value = 0;
            for (int j = 0; j < 8; j++)
            {
                int3 coord = base_coord + make_int3((j >> 2) & 1, (j >> 1) & 1, j & 1);
                interpolation_value += func(solver->pg.getCenter(coord)) * weights[j];
            }
            float guass_x[TRI_GAUSS_NUM][2] = TRI_GAUSS_XS;
            float guass_w[TRI_GAUSS_NUM] = TRI_GAUSS_WS;
            float3 dst_v[3] = {
                {vertices[particle.indices.x]}, {vertices[particle.indices.y]}, {vertices[particle.indices.z]}};
            float trg_jacobian = jacobian(dst_v);
            float ground_truth = 0;
            for (int i = 0; i < TRI_GAUSS_NUM; i++)
            {
                float3 v = local_to_global(guass_x[i][0], guass_x[i][1], dst_v);
                ground_truth += 0.5 * guass_w[i] * func(v) * trg_jacobian;
            }
            REQUIRE(interpolation_value == Approx(ground_truth).margin(1e-3));
        }
    }

    float3 center_offset = make_float3(RAND_SIGN, RAND_SIGN, RAND_SIGN) * 0.3;
    float3 center = make_float3(16, 16, 16) + center_offset;
    float3 near_test_offset = make_float3(RAND_SIGN, RAND_SIGN, RAND_SIGN) * (1.4 + RAND_F * 0.5);
    float3 near_test = make_float3(16, 16, 16) + near_test_offset;
    float3 far_test_offset = make_float3(RAND_SIGN, RAND_SIGN, RAND_SIGN) * (4.3 + RAND_F * 0.5);
    float3 far_test = make_float3(16, 16, 16) + far_test_offset;
    LOG("center:" << center);
    LOG("near_test:" << near_test);
    LOG("far_test:" << far_test);
    solver->clear();
    solver = empty_pppm(32);
    float frequency = 3000;
    float omega = 2 * M_PI * frequency;
    SineSource source(omega);

#define PRECOMPUTE_STEP 128
    CArr<float> particle_far_field(PRECOMPUTE_STEP);
    CArr<float> particle_far_field_from_solver(PRECOMPUTE_STEP);
    GArr3D<float> visual_data_far_field(PRECOMPUTE_STEP, 32, 32);

    add_small_triangles(solver, {center, far_test}, 0.1);
    vertices = solver->pg.vertices.cpu();
    particles = solver->pg.particles.cpu();
    solver->precompute_grid_cache();
    solver->precompute_particle_cache();
    particle_far_field.reset();
    particle_far_field_from_solver.reset();
    for (int i = 0; i < PRECOMPUTE_STEP; i++)
    {
        solver->solve_fdtd_far_with_cache();
        cuExecuteBlock(1, 2, set_signal_kernel, *solver, source);
        solver->update_particle_dirichlet();
        auto far_history = solver->particle_history.cpu();
        particle_far_field_from_solver[i] = far_history[0].dirichlet[solver->fdtd.t];
        far_history[0].dirichlet[solver->fdtd.t] = 0;
        particle_far_field[i] =
            solver->bem.laplace(vertices.data(), PairInfo(particles[1].indices, particles[0].indices),
                                far_history[1].neumann, far_history[1].dirichlet, solver->fdtd.t) +
            solver->bem.laplace(vertices.data(), PairInfo(particles[0].indices, particles[0].indices),
                                far_history[0].neumann, far_history[0].dirichlet, solver->fdtd.t);
        far_history.reset();
        far_history[0].dirichlet[solver->fdtd.t] = 1;
        float factor =
            1.0f / 2 - solver->bem.laplace(vertices.data(), PairInfo(particles[0].indices, particles[0].indices),
                                           far_history[0].neumann, far_history[0].dirichlet, solver->fdtd.t);
        particle_far_field[i] = particle_far_field[i] / factor;
        cuExecuteBlock(1, 2, set_signal_kernel, *solver, source);
        solver->solve_fdtd_near_with_cache();
        visual_data_far_field[i].assign(solver->far_field[i][15]);
        // printf("%d: far field: %e, %e\n", i, particle_far_field[i], particle_far_field_from_solver[i]);
    }
    write_to_txt("particle_far_field.txt", particle_far_field);
    write_to_txt("particle_far_field_from_solver.txt", particle_far_field_from_solver);
    // renderArray(RenderElement(visual_data_far_field, 2e10f, "far_field"));

    solver->clear();
    solver = empty_pppm(32);
    CArr<float> particle_near_field(PRECOMPUTE_STEP);
    CArr<float> particle_near_field_from_solver(PRECOMPUTE_STEP);

    add_small_triangles(solver, {center, near_test}, 0.1);
    vertices = solver->pg.vertices.cpu();
    particles = solver->pg.particles.cpu();
    solver->precompute_grid_cache();
    solver->precompute_particle_cache();
    particle_near_field.reset();
    particle_near_field_from_solver.reset();
    for (int i = 0; i < PRECOMPUTE_STEP; i++)
    {
        solver->solve_fdtd_far_with_cache();
        cuExecuteBlock(1, 2, set_signal_kernel, *solver, source);
        solver->update_particle_dirichlet();
        auto near_history = solver->particle_history.cpu();
        particle_near_field_from_solver[i] = near_history[0].dirichlet[solver->fdtd.t];
        near_history[0].dirichlet[solver->fdtd.t] = 0;
        particle_near_field[i] =
            solver->bem.laplace(vertices.data(), PairInfo(particles[1].indices, particles[0].indices),
                                near_history[1].neumann, near_history[1].dirichlet, solver->fdtd.t) +
            solver->bem.laplace(vertices.data(), PairInfo(particles[0].indices, particles[0].indices),
                                near_history[0].neumann, near_history[0].dirichlet, solver->fdtd.t);
        near_history.reset();
        near_history[0].dirichlet[solver->fdtd.t] = 1;
        float factor =
            1.0f / 2 - solver->bem.laplace(vertices.data(), PairInfo(particles[0].indices, particles[0].indices),
                                           near_history[0].neumann, near_history[0].dirichlet, solver->fdtd.t);
        particle_near_field[i] = particle_near_field[i] / factor;
        cuExecuteBlock(1, 2, set_signal_kernel, *solver, source);
        solver->solve_fdtd_near_with_cache();
        // printf("%d: near field: %e, %e\n", i, particle_near_field[i], particle_near_field_from_solver[i]);
    }
    write_to_txt("particle_near_field.txt", particle_near_field);
    write_to_txt("particle_near_field_from_solver.txt", particle_near_field_from_solver);
}
