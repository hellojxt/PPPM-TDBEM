#include "hip/hip_runtime.h"
#include <vector>
#include "array_writer.h"
#include "bem.h"
#include "case_generator.h"
#include "gui.h"
#include "macro.h"
#include "objIO.h"
#include "pppm.h"
#include "sound_source.h"
#include "visualize.h"
#include "window.h"

#define ALL_STEP 512
#define SET_DIRICHLET false

using namespace pppm;

__global__ void set_boundary_value(PPPMSolver pppm, SineSource sine, MonoPole mp)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= pppm.pg.triangles.size())
        return;
    auto &p = pppm.pg.triangles[i];
    int t = pppm.time_idx();
    float dt = pppm.dt();
    if (SET_DIRICHLET)
        pppm.dirichlet[i][t] = (mp.dirichlet(p.center) * sine(dt * t)).real();
    pppm.neumann[i][t] = (mp.neumann(p.center, p.normal) * sine(dt * t)).real();
}

int main()
{
    int res = 32;
    PPPMSolver *solver = empty_pppm(res);
    auto filename = ASSET_DIR + std::string("sphere3.obj");
    auto mesh = Mesh::loadOBJ(filename, true);
    mesh.stretch_to(solver->size().x / 4.0f);
    LOG("stretch to " << mesh.get_scale())
    mesh.move_to(solver->center());

    solver->set_mesh(mesh.vertices, mesh.triangles);
    RenderElement re(solver->pg, "PPPM");
    int x_idx = res / 6;
    int y_idx = res / 2;
    int z_idx = res / 2;

    re.set_params(make_int3(0, 0, z_idx), ALL_STEP, 1.0f);

    auto sine = SineSource(2 * PI * 3000);
    float wave_number = sine.omega / AIR_WAVE_SPEED;
    LOG("wave number: " << wave_number)
    auto mp = MonoPole(solver->center(), wave_number);

    TICK(solve_with_cache)
    for (int i = 0; i < ALL_STEP; i++)
    {
        solver->pg.fdtd.step();
        solver->solve_fdtd_far();
        cuExecute(solver->pg.triangles.size(), set_boundary_value, *solver, sine, mp);
        if (!SET_DIRICHLET)
            solver->update_dirichlet();
        solver->solve_fdtd_near();
        re.assign(i, solver->pg.fdtd.grids[i]);
    }
    TOCK(solve_with_cache)

    TDBEM &bem = solver->bem;
    auto vertices = mesh.vertices;
    auto paticles = solver->pg.triangles.cpu();
    float3 trg_pos = solver->pg.getCenter(x_idx, y_idx, z_idx);
    cpx bem_sum = 0;
    for (int p_id = 0; p_id < paticles.size(); p_id++)
    {
        auto &p = paticles[p_id];
        auto pair_info = PairInfo(p.indices, trg_pos);
        bem_sum += bem.helmholtz(vertices.data(), pair_info, mp.neumann(p.center, p.normal), mp.dirichlet(p.center),
                                 wave_number);
    }

    auto solver_signal = re.get_time_siganl(y_idx, x_idx).cpu();
    CArr<float> helmholtz_result(ALL_STEP);
    for (int i = 0; i < ALL_STEP; i++)
        helmholtz_result[i] = (bem_sum * sine(solver->dt() * i)).real();
    CArr<float> analytic_result(ALL_STEP);
    for (int i = 0; i < ALL_STEP; i++)
        analytic_result[i] = (mp.dirichlet(trg_pos) * sine(solver->dt() * i)).real();

    LOG("bem sum: " << bem_sum)
    LOG("analytic weight: " << mp.dirichlet(trg_pos))
    write_to_txt("pppm_signal.txt", solver_signal);
    write_to_txt("helmholtz_signal.txt", helmholtz_result);
    write_to_txt("analytic_signal.txt", analytic_result);
    re.update_mesh();
    // re.write_image(ALL_STEP / 2, "pppm.png");
    // renderArray(re);
}
