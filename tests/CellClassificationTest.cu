#include "hip/hip_runtime.h"
#include "case_generator.h"
#include "ghost_cell.h"
#include "macro.h"
#include "objIO.h"
#include "visualize.h"
using namespace pppm;
struct view_transformer
{
        CGPU_FUNC float operator()(CellInfo &x) const
        {
            if (x.type == SOLID)
                return 0.0f;  // green for solid
            if (x.type == GHOST)
                return 1.0f;  // red for ghost
            if (x.type == AIR)
                return -1.0f;  // blue for air
            return 0.5f;       // yellow unknown
        }
};

void view_cell_data(GhostCellSolver *solver)
{
    GArr3D<float> view_data;
    view_data.resize(solver->cell_data.size);
    thrust::transform(thrust::device, solver->cell_data.begin(), solver->cell_data.end(), view_data.begin(),
                      view_transformer());

    RenderElement re(solver->grid, "distance");
    re.set_params(make_int3(0, 0, 32), 1, 1.0f);
    re.assign(0, view_data);
    re.update_mesh();
    re.write_image(0, EXP_DIR + std::string("test/cell_classification.png"));
}

int main()
{
    GhostCellSolver *solver = empty_ghost_cell_solver(64);
    auto filename = ASSET_DIR + std::string("sphere.obj");
    auto mesh = Mesh::loadOBJ(filename, true);
    mesh.stretch_to(solver->size().x / 3.0f);
    mesh.move_to(solver->center());

    solver->set_mesh(mesh.vertices, mesh.triangles);
    solver->precompute_cell_data();

    view_cell_data(solver);
}