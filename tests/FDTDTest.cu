#include "hip/hip_runtime.h"
#include "gui.h"
#include "window.h"
#include "objIO.h"
#include "fdtd.h"
#include "sound_source.h"

using namespace pppm;

__global__ void set_center_signal(FDTD fdtd, SineSource s){
    int3 center = make_int3(fdtd.res / 2, fdtd.res / 2, fdtd.res / 2);
    fdtd.grids[fdtd.t - 1](center) = s((fdtd.t - 1) * fdtd.dt).real(); // ftdt.t is the next time step
}

int main()
{
	GUI gui;
	CudaRender render;

    int res = 51;
	int step_num = 300;
	float dl = 0.005;
	float dt = 1.0f / 150000;

	GArr3D<float> data;
    data.resize(step_num, res, res);
	FDTD fdtd;
	fdtd.init(res, dl, dt);
    SineSource s(5000 * 2 * M_PI);

	for (int i = 0; i < step_num; i++)
	{
		fdtd.step();
        cuExecuteBlock(1, 1, set_center_signal, fdtd, s);
		data[i].assign(fdtd.grids[i][25]);
	}
	render.setData(data, 0.02f);
	gui.append(&render);
	gui.start();
}