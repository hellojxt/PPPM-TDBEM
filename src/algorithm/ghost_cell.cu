#include "hip/hip_runtime.h"
#include "ghost_cell.h"
#include "ghost_cell_util.h"
#include "macro.h"
namespace pppm
{

CGPU_FUNC inline int3 neighbor_idx_to_coord(int idx)
{
    return make_int3(idx % 2, (idx / 2) % 2, idx / 4);
}

// normalize the coordinate to the range [-1, 1]^3
GPU_FUNC inline float3 get_normalized_coord(float3 coord, int3 base_coord, GhostCellSolver &solver)
{
    float3 base_point = solver.grid.getCenter(base_coord);
    return (coord - base_point) / solver.grid_size() * 2 + make_float3(-1, -1, -1);
}

// return the base coordinate of the 8 interpolation points of the reflect point
GPU_FUNC inline int3 get_base_coord_for_reflect(CellInfo ghost_cell, GhostCellSolver &solver)
{
    float grid_size = solver.grid_size();
    float3 reflect_point = ghost_cell.reflect_point;
    int3 base_coord = make_int3((reflect_point - solver.grid.min_pos) / grid_size - 0.5f);
#ifdef MEMORY_CHECK
    float3 base_point = solver.grid.getCenter(base_coord);
    float3 offset = reflect_point - base_point;
    float eps = grid_size * 1e-3;
    // if (offset.x < -eps || offset.y < -eps || offset.z < -eps || offset.x > grid_size + eps ||
    //     offset.y > grid_size + eps || offset.z > grid_size + eps)
    // {
    //     float3 tmp = (reflect_point - solver.grid.min_pos) / grid_size;
    //     printf("%f, %f, %f\n", tmp.x, tmp.y, tmp.z);
    //     printf("t = %d\n", solver.grid.fdtd.t);
    //     printf("offset: %f %f %f\n", offset.x, offset.y, offset.z);
    //     printf("base_point: %f %f %f\n", base_point.x, base_point.y, base_point.z);
    //     printf("reflect_point: %f %f %f\n", reflect_point.x, reflect_point.y, reflect_point.z);
    //     printf("grid_size: %f\n", grid_size);
    //     printf("base_coord: %d %d %d\n", base_coord.x, base_coord.y, base_coord.z);
    //     printf("grid_dim: %d\n", solver.grid.grid_dim);
    //     printf("min_pos: %f %f %f\n", solver.grid.min_pos.x, solver.grid.min_pos.y, solver.grid.min_pos.z);
    // }

    assert(offset.x >= -eps && offset.y >= -eps && offset.z >= -eps && offset.x <= grid_size + eps &&
           offset.y <= grid_size + eps && offset.z <= grid_size + eps);

#endif
    return base_coord;
}

__global__ void get_fresh_cell_list(GhostCellSolver solver)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    // here we assume grid_dim is (a, a, a).
    int grid_dim = solver.cell_data.size.x;
    if (x < 0 || x >= grid_dim || y < 0 || y >= grid_dim || z < 0 || z >= grid_dim)
        return;
    auto type = solver.cell_data(x, y, z).type;
    auto old_type = solver.cell_data_old(x, y, z).type;
    auto index = solver.cell_data.index(x, y, z);
    solver.fresh_cell_list[index].coord = make_int3(x, y, z);
    if (type != CellType::AIR)
    {
        solver.fresh_cell_list[index].is_fresh = true;
    }
    else
    {
        solver.fresh_cell_list[index].is_fresh = false;
    }
}

__global__ void solve_fresh_history(GhostCellSolver solver)
{
    int list_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (list_idx >= solver.fresh_cell_list.size())
        return;
    int3 coord = solver.fresh_cell_list[list_idx].coord;
    auto &cell = solver.cell_data(coord);
    solver.grid.fdtd.grids[solver.grid.fdtd.t](coord) = 1e10;
    if (cell.nearest_particle_idx >= solver.neuuman_data.size())
        printf("cell.nearest_particle_idx: %d, neuumann_data.size(): %d\n", cell.nearest_particle_idx,
               solver.neuuman_data.size());
    // float accs[2] = {solver.neuuman_data[cell.nearest_particle_idx],
    //                  solver.neuuman_data_old[cell.nearest_particle_idx]};
    // int ts[2] = {solver.grid.fdtd.t, solver.grid.fdtd.t - 1};
    // float3 xb = cell.nearst_point;
    // float3 xf = solver.grid.getCenter(coord);
    // float3 xr = xb + 2 * (xf - xb);
    // int3 neighbor_list[8];
    // float neighbor_coeff_list[8];
    // int3 base_coord = solver.grid.getGridBaseCoord(xr);
    // for (int dx = 0; dx < 2; dx++)
    //     for (int dy = 0; dy < 2; dy++)
    //         for (int dz = 0; dz < 2; dz++)
    //         {
    //             int3 neighbor_coord = base_coord + make_int3(dx, dy, dz);
    //             int idx = dx * 4 + dy * 2 + dz;
    //             neighbor_list[idx] = neighbor_coord;
    //             if (solver.cell_data_old(neighbor_coord).type == AIR && solver.cell_data(neighbor_coord).type == AIR)
    //             {
    //                 float3 neighor_center = solver.grid.getCenter(neighbor_coord);
    //                 float dist = length(neighor_center - xr);
    //                 neighbor_coeff_list[idx] = 1.0 / (dist * dist);
    //             }
    //             else
    //             {
    //                 neighbor_coeff_list[idx] = 0;
    //             }
    //         }

    // float sum = 0;
    // for (int i = 0; i < 8; i++)
    // {
    //     sum += neighbor_coeff_list[i];
    // }
    // for (int i = 0; i < 8; i++)
    // {
    //     neighbor_coeff_list[i] /= sum;
    // }

    // #pragma unroll
    //     for (int i = 0; i < 2; i++)
    //     {
    //         float acc = accs[i];
    //         int t = ts[i];
    //         float pr = 0;
    //         for (int j = 0; j < 8; j++)
    //         {
    //             int3 neighbor_coord = neighbor_list[j];
    //             float coeff = neighbor_coeff_list[j];
    //             pr += coeff * solver.grid.fdtd.grids[t](neighbor_coord);
    //         }
    //         float pf = pr - AIR_DENSITY * acc * length(xf - xr);
    //         solver.grid.fdtd.grids[t](coord) = pf;
    //     }
}

void GhostCellSolver::fill_in_fresh_cell(bool log_time)
{
    START_TIME(log_time)
    cuExecute3D(dim3(grid.grid_dim, grid.grid_dim, grid.grid_dim), get_fresh_cell_list, *this);
    fresh_cell_list.remove_zeros();
    if (log_time)
    {
        LOG("Fresh cell: " << fresh_cell_list.size())
    }
    cuExecute(fresh_cell_list.size(), solve_fresh_history, *this);
    LOG_TIME("Fill in fresh cell")
}

__global__ void construct_ghost_cell_list(GhostCellSolver solver)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    // here we assume grid_dim is (a, a, a).
    int grid_dim = solver.cell_data.size.x;
    if (x < 0 || x >= grid_dim || y < 0 || y >= grid_dim || z < 0 || z >= grid_dim)
        return;
    CellInfo cell = solver.cell_data(x, y, z);
    if (cell.type == CellType::GHOST)
    {
        solver.ghost_cells[cell.ghost_idx] = make_int3(x, y, z);
    }
}

void GhostCellSolver::precompute_cell_data(bool log_time)
{
    START_TIME(log_time)
    ghost_cell_num = fill_cell_data(grid, cell_data, (condition_number_threshold == 0));
    ghost_cells.resize(ghost_cell_num);
    cuExecute3D(dim3(grid.grid_dim, grid.grid_dim, grid.grid_dim), construct_ghost_cell_list, *this);
    if (ghost_cell_num <= 0)
        LOG_ERROR("No ghost cell found!");
    LOG_TIME("Precompute Cell Data")
};

__global__ void construct_phi_matrix_kernel(GArr3D<float> phi, GhostCellSolver solver)
{
    int ghost_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (ghost_idx >= solver.ghost_cell_num)
        return;
    int3 ghost_cell_coord = solver.ghost_cells[ghost_idx];
    CellInfo ghost_cell = solver.cell_data(ghost_cell_coord);
    float3 normal = normalize(ghost_cell.reflect_point - ghost_cell.nearst_point);
    int3 base_coord = get_base_coord_for_reflect(ghost_cell, solver);

    for (int i = 0; i < GHOST_CELL_NEIGHBOR_NUM; i++)
    {
        int3 dcoord = neighbor_idx_to_coord(i);
        int3 neighbor_coord = base_coord + dcoord;
        bool is_self = (neighbor_coord.x == ghost_cell_coord.x && neighbor_coord.y == ghost_cell_coord.y &&
                        neighbor_coord.z == ghost_cell_coord.z);
        if (is_self)
        {
            // dn phi = normal dot D phi
            float3 coord = get_normalized_coord(ghost_cell.nearst_point, base_coord, solver);
            phi(ghost_idx, i, 0) = dot(normal, make_float3(coord.y * coord.z, coord.x * coord.z, coord.x * coord.y));
            phi(ghost_idx, i, 1) = dot(normal, make_float3(coord.y, coord.x, 0));
            phi(ghost_idx, i, 2) = dot(normal, make_float3(0, coord.z, coord.y));
            phi(ghost_idx, i, 3) = dot(normal, make_float3(coord.z, 0, coord.x));
            phi(ghost_idx, i, 4) = dot(normal, make_float3(1, 0, 0));
            phi(ghost_idx, i, 5) = dot(normal, make_float3(0, 1, 0));
            phi(ghost_idx, i, 6) = dot(normal, make_float3(0, 0, 1));
            phi(ghost_idx, i, 7) = 0;
        }
        else
        {
            // phi[ghost_idx][i] = [xyz, xy, xz, yz, x, y, z, 1];
            float3 coord = make_float3(dcoord * 2 - 1);
            phi(ghost_idx, i, 0) = coord.x * coord.y * coord.z;
            phi(ghost_idx, i, 1) = coord.x * coord.y;
            phi(ghost_idx, i, 2) = coord.y * coord.z;
            phi(ghost_idx, i, 3) = coord.x * coord.z;
            phi(ghost_idx, i, 4) = coord.x;
            phi(ghost_idx, i, 5) = coord.y;
            phi(ghost_idx, i, 6) = coord.z;
            phi(ghost_idx, i, 7) = 1;
        }
    }
}

__global__ void precompute_p_weight_kernel(SVDResult svd_result, GhostCellSolver solver)
{
    int ghost_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (ghost_idx >= solver.ghost_cell_num)
        return;
    float max_singular_value = 0;
    float min_singular_value = MAX_FLOAT;
    for (int i = 0; i < GHOST_CELL_NEIGHBOR_NUM; i++)
    {
        float singular_value = svd_result.S(ghost_idx, i);
        if (singular_value > max_singular_value)
            max_singular_value = singular_value;
        if (singular_value < min_singular_value)
            min_singular_value = singular_value;
    }
    float condition_num = max_singular_value / min_singular_value;

    // printf("condition_num: %f , threshold: %f\n", condition_num, solver.condition_number_threshold);
    if (condition_num > solver.condition_number_threshold)
    {
        solver.ghost_order[ghost_idx] = AccuracyOrder::FIRST_ORDER;
        for (int i = 0; i < GHOST_CELL_NEIGHBOR_NUM; i++)
        {
            solver.p_weight(ghost_idx, i) = 0;
        }
    }
    else
    {
        solver.ghost_order[ghost_idx] = AccuracyOrder::SECOND_ORDER;
        int3 ghost_cell_coord = solver.ghost_cells[ghost_idx];
        auto ghost_cell = solver.cell_data(ghost_cell_coord);
        int3 base_coord = get_base_coord_for_reflect(ghost_cell, solver);
        float3 r = get_normalized_coord(ghost_cell.reflect_point, base_coord, solver);
        // printf("r: %f %f %f\n", r.x, r.y, r.z);
        float phi_r[GHOST_CELL_NEIGHBOR_NUM] = {r.x * r.y * r.z, r.x * r.y, r.y * r.z, r.x * r.z, r.x, r.y, r.z, 1};
        // p_weight = inv_A.T * phi_r
        for (int i = 0; i < GHOST_CELL_NEIGHBOR_NUM; i++)
        {
            float sum = 0;
            for (int j = 0; j < GHOST_CELL_NEIGHBOR_NUM; j++)
            {
                sum += svd_result.inv_A(ghost_idx, j, i) * phi_r[j];
            }
            if (isnan(sum))
            {
                solver.ghost_order[ghost_idx] = AccuracyOrder::FIRST_ORDER;
                for (int k = 0; k < GHOST_CELL_NEIGHBOR_NUM; k++)
                {
                    solver.p_weight(ghost_idx, i) = 0;
                }
                return;
            }
            solver.p_weight(ghost_idx, i) = sum;
        }
    }
}

template <bool CONSTRUCT_MATRIX = true, bool CONSTRUCT_RHS = true>
__global__ void construct_equation_kernel(GhostCellSolver solver)
{
    int ghost_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (ghost_idx >= solver.ghost_cell_num)
        return;
    auto acc_order = solver.ghost_order[ghost_idx];
    int offset = ghost_idx * (GHOST_CELL_NEIGHBOR_NUM + 1);
    int3 ghost_cell_coord = solver.ghost_cells[ghost_idx];
    if (acc_order == AccuracyOrder::FIRST_ORDER)
    {
        if (CONSTRUCT_MATRIX)
        {
            solver.A.rows[offset] = ghost_idx;
            solver.A.cols[offset] = ghost_idx;
            solver.A.vals[offset] = 1;
        }
        if (CONSTRUCT_RHS)
        {
            int3 dcoord[6] = {make_int3(1, 0, 0),  make_int3(-1, 0, 0), make_int3(0, 1, 0),
                              make_int3(0, -1, 0), make_int3(0, 0, 1),  make_int3(0, 0, -1)};
            auto cell = solver.cell_data(ghost_cell_coord);
            int neighbor_num = 0;
            solver.b[ghost_idx] = 0;
            for (int i = 0; i < 6; i++)
            {
                int3 coord = ghost_cell_coord + dcoord[i];
                if (solver.cell_data(coord).type == AIR)
                {
                    solver.b[ghost_idx] +=
                        -solver.neuuman_data[cell.nearest_particle_idx] * solver.grid_size() +
                        solver.grid.fdtd.grids[solver.grid.fdtd.t](coord);  // p_g - p_n = l * rho * a_n
                    neighbor_num++;
                }
            }
            solver.b[ghost_idx] /= neighbor_num;
        }
    }
    else if (acc_order == AccuracyOrder::SECOND_ORDER)
    {
        auto ghost_cell = solver.cell_data(ghost_cell_coord);
        int3 base_coord = get_base_coord_for_reflect(ghost_cell, solver);
        float b_value = 0;
        if (CONSTRUCT_RHS)
            b_value += -solver.neuuman_data[ghost_cell.nearest_particle_idx] * ghost_cell.nearst_distance * 2;
        for (int i = 0; i < GHOST_CELL_NEIGHBOR_NUM; i++)
        {
            int3 dcoord = neighbor_idx_to_coord(i);
            int3 neighbor_coord = base_coord + dcoord;
            auto neighbor_cell = solver.cell_data(neighbor_coord);
            bool is_self = (neighbor_cell.ghost_idx == ghost_idx);
            if (is_self)  // ghost cell self, p = -rho*an(nearst_point)
            {
                if (CONSTRUCT_RHS)
                {
                    float scale_factor =
                        solver.grid_size() / 2;  // correction factor as stencils are transformed to the [−1, 1]^3
                    b_value += solver.p_weight(ghost_idx, i) *
                               (scale_factor * solver.neuuman_data[ghost_cell.nearest_particle_idx]);
                }
            }
            else if (neighbor_cell.type == GHOST)  // other ghost cell, add matrix element
            {
                if (CONSTRUCT_MATRIX)
                {
                    solver.A.rows[offset + i] = ghost_idx;
                    solver.A.cols[offset + i] = neighbor_cell.ghost_idx;
                    // solver.A.vals[offset + i] = 0;
                    solver.A.vals[offset + i] = -solver.p_weight(ghost_idx, i);
                    // printf("A(%d, %d) = %f\n", ghost_idx, neighbor_cell.ghost_idx, solver.p_weight(ghost_idx, i));
                }
            }
            else  // neighbor cell is air
            {
                if (CONSTRUCT_RHS)
                {
                    b_value +=
                        solver.p_weight(ghost_idx, i) * solver.grid.fdtd.grids[solver.grid.fdtd.t](neighbor_coord);
                }
            }
        }
        if (CONSTRUCT_RHS)
        {
            solver.b[ghost_idx] = b_value;
        }

        if (CONSTRUCT_MATRIX)
        {
            solver.A.rows[offset + GHOST_CELL_NEIGHBOR_NUM] = ghost_idx;
            solver.A.cols[offset + GHOST_CELL_NEIGHBOR_NUM] = ghost_idx;
            solver.A.vals[offset + GHOST_CELL_NEIGHBOR_NUM] = 1;
        }
    }
}

void GhostCellSolver::precompute_ghost_matrix(bool log_time)
{
    START_TIME(log_time)
    b.resize(ghost_cell_num);
    if (condition_number_threshold > 0.0f)
    {
        A.resize(ghost_cell_num, ghost_cell_num, ghost_cell_num * (GHOST_CELL_NEIGHBOR_NUM + 1));
        A.reset();  // set A to zero matrix
        x.resize(ghost_cell_num);
        p_weight.resize(ghost_cell_num, GHOST_CELL_NEIGHBOR_NUM);
        GArr3D<float> phi;
        phi.resize(ghost_cell_num, GHOST_CELL_NEIGHBOR_NUM, GHOST_CELL_NEIGHBOR_NUM);
        cuExecute(ghost_cell_num, construct_phi_matrix_kernel, phi, *this);
        LOG_TIME("Construct phi matrix")
        auto svd_result = cusolver_svd(phi);
        svd_result.solve_inverse();
        LOG_TIME("SVD")
        ghost_order.resize(ghost_cell_num);
        cuExecute(ghost_cell_num, precompute_p_weight_kernel, svd_result, *this);
        LOG_TIME("Precompute p weight")
        auto construct_matrix_kernel = construct_equation_kernel<true, false>;
        cuExecute(ghost_cell_num, construct_matrix_kernel, *this);
        A.eliminate_zeros();
        A.sort_by_row();
        linear_solver.set_coo_matrix(A);
        LOG_TIME("Construct matrix A")
        phi.clear();
        svd_result.clear();
    }
    else
    {
        ghost_order.resize(ghost_cell_num);
        ghost_order.reset();
    }
}

__global__ void update_ghost_cell_kernel(GArr<float> x, GhostCellSolver solver)
{
    int ghost_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (ghost_idx >= solver.ghost_cell_num)
        return;
    int3 ghost_cell_coord = solver.ghost_cells[ghost_idx];
    // if (abs(x[ghost_idx]) > 1e3)
    // {
    //     printf("t = %d\n", solver.grid.fdtd.t);
    //     printf("x[%d] = %f at (%d, %d, %d)\n", ghost_idx, x[ghost_idx], ghost_cell_coord.x, ghost_cell_coord.y,
    //            ghost_cell_coord.z);
    // }
    solver.grid.fdtd.grids[solver.grid.fdtd.t](ghost_cell_coord) = x[ghost_idx];
}

void GhostCellSolver::solve_ghost_cell(bool log_time)
{
    START_TIME(log_time)
    b.reset();
    auto construct_rhs_kernel = construct_equation_kernel<false, true>;
    cuExecute(ghost_cell_num, construct_rhs_kernel, *this);
    LOG_TIME("Construct rhs b")
    if (condition_number_threshold == 0.0f)
    {
        cuExecute(ghost_cell_num, update_ghost_cell_kernel, b, *this);
    }
    else
    {
        linear_solver.solve(b, x);
        cuExecute(ghost_cell_num, update_ghost_cell_kernel, x, *this);
        LOG_TIME("Solve equation for ghost cell")
    }
}

__global__ void set_solid_cell_zero_kernel(FDTD fdtd, GArr3D<CellInfo> cell_data)
{
    int3 coord = make_int3(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y,
                           blockIdx.z * blockDim.z + threadIdx.z);
    if (coord.x >= fdtd.res || coord.y >= fdtd.res || coord.z >= fdtd.res)
        return;
    if (cell_data(coord).type == SOLID)
        fdtd.grids[fdtd.t](coord) = 0;
}

void GhostCellSolver::set_solid_cell_zero()
{
    cuExecute3D(dim3(grid.grid_dim, grid.grid_dim, grid.grid_dim), set_solid_cell_zero_kernel, grid.fdtd, cell_data);
}

}  // namespace pppm