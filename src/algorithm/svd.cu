#include "hip/hip_runtime.h"
#include "array3D.h"
#include "svd.h"
#include <cmath>
#include <functional>
#include <iostream>
#include <random>
#include <stdexcept>
#include <string>

namespace pppm
{
// CUDA API error checking
#define CUDA_CHECK(err)                                                   \
    do                                                                    \
    {                                                                     \
        hipError_t err_ = (err);                                         \
        if (err_ != hipSuccess)                                          \
        {                                                                 \
            printf("CUDA error %d at %s:%d\n", err_, __FILE__, __LINE__); \
            throw std::runtime_error("CUDA error");                       \
        }                                                                 \
    } while (0)

// cusolver API error checking
#define CUSOLVER_CHECK(err)                                                   \
    do                                                                        \
    {                                                                         \
        hipsolverStatus_t err_ = (err);                                        \
        if (err_ != HIPSOLVER_STATUS_SUCCESS)                                  \
        {                                                                     \
            printf("cusolver error %d at %s:%d\n", err_, __FILE__, __LINE__); \
            throw std::runtime_error("cusolver error");                       \
        }                                                                     \
    } while (0)

SVDResult cusolver_svd(GArr3D<float> A)
{
    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;
    int batchSize = A.batchs;
    int m = A.rows;
    int n = A.cols;
    if (m != n)
    {
        throw std::runtime_error("cusolver_svd: m != n");
    }
    int lda = m;
    int ldu = m;
    int ldv = m;
    int rank = m;
    long long int strideA = static_cast<long long int>(lda * m);
    long long int strideS = m;
    long long int strideU = static_cast<long long int>(ldu * m);
    long long int strideV = static_cast<long long int>(ldv * m);

    float *d_A = A.begin();
    GArr2D<float> S(A.batchs, m); /* singular values */
    float *d_S = S.begin();
    GArr3D<float> U(A.batchs, m, m); /* left singular vectors */
    float *d_U = U.begin();
    GArr3D<float> V(A.batchs, m, m); /* right singular vectors */
    float *d_V = V.begin();
    GArr<int> info(A.batchs); /* error info */
    int *d_info = info.begin();

    int lwork = 0;           /* size of workspace */
    float *d_work = nullptr; /* device workspace for getrf */

    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; /* compute eigenvectors */

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    /* step 3: query working space of SVD */
    CUSOLVER_CHECK(hipsolverDnSgesvdaStridedBatched_bufferSize(
        cusolverH, jobz,  /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
                          /* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors */
        rank,             /* number of singular values */
        m,                /* nubmer of rows of Aj, 0 <= m */
        n,                /* number of columns of Aj, 0 <= n  */
        d_A,              /* Aj is m-by-n */
        lda,              /* leading dimension of Aj */
        strideA,          /* >= lda*n */
        d_S,              /* Sj is rank-by-1, singular values in descending order */
        strideS,          /* >= rank */
        d_U,              /* Uj is m-by-rank */
        ldu,              /* leading dimension of Uj, ldu >= max(1,m) */
        strideU,          /* >= ldu*rank */
        d_V,              /* Vj is n-by-rank */
        ldv,              /* leading dimension of Vj, ldv >= max(1,n) */
        strideV,          /* >= ldv*rank */
        &lwork, batchSize /* number of matrices */
        ));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(float) * lwork));
    std::vector<double> RnrmF(batchSize, 0); /* residual norm */
    /* step 4: compute SVD */
    CUSOLVER_CHECK(hipsolverDnSgesvdaStridedBatched(
        cusolverH, jobz, /* HIPSOLVER_EIG_MODE_NOVECTOR: compute singular values only */
                         /* HIPSOLVER_EIG_MODE_VECTOR: compute singular value and singular vectors */
        rank,            /* number of singular values */
        m,               /* nubmer of rows of Aj, 0 <= m */
        n,               /* number of columns of Aj, 0 <= n  */
        d_A,             /* Aj is m-by-n */
        lda,             /* leading dimension of Aj */
        strideA,         /* >= lda*n */
        d_S,             /* Sj is rank-by-1 */
                         /* the singular values in descending order */
        strideS,         /* >= rank */
        d_U,             /* Uj is m-by-rank */
        ldu,             /* leading dimension of Uj, ldu >= max(1,m) */
        strideU,         /* >= ldu*rank */
        d_V,             /* Vj is n-by-rank */
        ldv,             /* leading dimension of Vj, ldv >= max(1,n) */
        strideV,         /* >= ldv*rank */
        d_work, lwork, d_info, RnrmF.data(), batchSize /* number of matrices */
        ));

    CUDA_CHECK(hipStreamSynchronize(stream));
    /* free resources */
    CUDA_CHECK(hipFree(d_work));
    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
    CUDA_CHECK(hipStreamDestroy(stream));
    return SVDResult(S, U, V, info);
}

void __global__ inverse_from_svd_kernel(GArr2D<float> S, GArr3D<float> U, GArr3D<float> V, GArr3D<float> inv_A)
{
    int batch = blockIdx.x;
    int row = threadIdx.x;
    int col = threadIdx.y;
    float sum = 0;
    // A^T = (U^T) * (S^T) * (V^T)^T
    // A = V^T * S * U
    // A^-1 = U^T * S^-1 * V
    for (int i = 0; i < U.rows; i++)
    {
        float s = S(batch, i);
        if (s > 1e-6)
        {
            s = 1 / s;
        }
        sum += U(batch, i, row) * s * V(batch, i, col);
    }
    inv_A(batch, row, col) = sum;
}

GArr3D<float> inverse_from_svd(GArr2D<float> S, GArr3D<float> U, GArr3D<float> V)
{
    GArr3D<float> inv_A(U.batchs, U.rows, U.cols);
    inverse_from_svd_kernel<<<U.batchs, dim3(U.rows, U.cols)>>>(S, U, V, inv_A);
    cuSynchronize();
    return inv_A;
}

};  // namespace pppm
