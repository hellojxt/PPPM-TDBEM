#include "hip/hip_runtime.h"
#include "Object.h"
#include "hip/hip_vector_types.h"
#include <fstream>
namespace pppm
{
__device__ inline float3 rotate(const float4 q, const float3 v)
{
    float3 u = make_float3(q.x, q.y, q.z);
    float s = q.w;
    return 2.0f * dot(u, v) * u + (s * s - dot(u, u)) * v + 2.0f * s * cross(u, v);
}

__global__ void Transform(GArr<float3> vertices, GArr<float3> standard_vertices, float3 translation, float4 rotation)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= vertices.size())
        return;
    vertices[id] = rotate(rotation, standard_vertices[id]) + translation;
    return;
}

__global__ void Fill(float* arr, float num, size_t size)
{
    int id = (threadIdx.x + blockIdx.x * blockDim.x) * 64;
    for(int i = 0; i < 64 && id + i < size; i++)
    {
        arr[id + i] = num;
    }
    return;
}

std::pair<CArr<int3>, CArr<float3>> FindAllSurfaces(CArr<int4> &tetrahedrons, CArr<float3> &tetraVerts)
{
    CArr<int3> surfaceTriangles;
    CArr<float3> surfaceNorms;

    struct iVec3Hash
    {
            int operator()(const int3 &vec) const { return (vec.x << 20) + (vec.y << 10) + vec.z; }
    };
    struct iVec3Eq
    {
            bool operator()(const int3 &vec1, const int3 &vec2) const
            {
                return vec1.x == vec2.x && vec1.y == vec2.y && vec1.z == vec2.z;
            }
    };
    struct TriInfo
    {
            int cnt = 0;
            int tetID = 0;
            int exceptVertID = 0;
    };

    std::unordered_map<int3, TriInfo, iVec3Hash, iVec3Eq> candidateTriangles;
    int3 currTri;
    int int3::*int3Members[] = {&int3::x, &int3::y, &int3::z};
    int int4::*int4Members[] = {&int4::x, &int4::y, &int4::z, &int4::w};

    for (int i = 0, size = tetrahedrons.size(); i < size; i++)
    {
        int4 &currVertIDs = tetrahedrons[i];
        for (int i0 = 0; i0 < 4; i0++)
        {
            for (int j = 0, k = 0; j < 4; j++)
            {
                if (j == i0)
                    continue;
                currTri.*(int3Members[k++]) = currVertIDs.*(int4Members[j]);
            }
            auto &currInfo = candidateTriangles[currTri];
            currInfo.cnt++;
            currInfo.tetID = i;
            currInfo.exceptVertID = i0;
        }
    }

    for (auto &candidateTriangle : candidateTriangles)
    {
        if (candidateTriangle.second.cnt != 1)
            continue;

        int3 currTri = candidateTriangle.first;

        int4 &currTet = tetrahedrons[candidateTriangle.second.tetID];
        int exceptVertID = candidateTriangle.second.exceptVertID;

        float3 center = (tetraVerts[currTri.x] + tetraVerts[currTri.y] + tetraVerts[currTri.z]) / 3;
        float3 exceptVec = tetraVerts[currTet.*(int4Members[exceptVertID])] - center;
        float3 e1 = tetraVerts[currTri.y] - tetraVerts[currTri.x], e2 = tetraVerts[currTri.z] - tetraVerts[currTri.x];
        float3 normVec = normalize(cross(e1, e2));
        if (dot(normVec, exceptVec) > 0)
        {
            std::swap(currTri.y, currTri.z);
            normVec = -normVec;
        }
        surfaceTriangles.pushBack(currTri);
        surfaceNorms.pushBack(normVec);
    }

    return {surfaceTriangles, surfaceNorms};
}

void Object::LoadTetMesh_(const std::string &vertsPath, const std::string &tetPath,
                          GArr<float3> &tetVertices, GArr<int3> &tetSurfaces, 
                          GArr<float3> &tetSurfaceNorms)
{
    std::ifstream f_verts(vertsPath);
    CArr<float3> cpuTetVertices;
    float3 vert;
    if (!f_verts.good())
    {
        LOG_ERROR("Fail to load tet mesh file.");
        std::exit(EXIT_FAILURE);
    }
    std::string line;
    while (getline(f_verts, line))
    {
        if (line.empty())
            continue;
        std::istringstream iss(line);
        iss >> vert.x >> vert.y >> vert.z;
        cpuTetVertices.pushBack(vert);
    }
    f_verts.close();

    std::ifstream f_tet(tetPath);
    CArr<int4> tetrahedrons;
    float4 tet;
    if (!f_tet.good())
    {
        LOG_ERROR("Fail to load tet mesh file.");
        std::exit(EXIT_FAILURE);
    }
    while (getline(f_tet, line))
    {
        if (line.empty())
            continue;
        std::istringstream iss(line);
        iss >> tet.x >> tet.y >> tet.z >> tet.w;
        int idxs[4] = {F2I(tet.x), F2I(tet.y), F2I(tet.z), F2I(tet.w)};
        std::sort(idxs, idxs + 4);
        tetrahedrons.pushBack(make_int4(idxs[0], idxs[1], idxs[2], idxs[3]));
    }

    tetVertices.assign(cpuTetVertices);
    auto [surfaceTris, surfaceNorms] = FindAllSurfaces(tetrahedrons, cpuTetVertices);
    tetSurfaces.assign(surfaceTris);
    tetSurfaceNorms.assign(surfaceNorms);
    return;
}

void Object::LoadMotion_(const std::string &path, CArr<float3> &translations,
                         CArr<float4> &rotations, CArr<float>& frameTime)
{
    std::ifstream fin(path);

    float currTime = 0;
    float3 currTranslation;
    float4 currRotation;
    if (!fin.good())
    {
        LOG_ERROR("Fail to load displacement file.\n");
        std::exit(EXIT_FAILURE);
    }
    std::string line;
    while (getline(fin, line))
    {
        if (line.empty())
            continue;
        std::istringstream iss(line);
        iss >> currTime >> currTranslation.x >> currTranslation.y >> currTranslation.z >> currRotation.x >>
            currRotation.y >> currRotation.z >> currRotation.w;
        translations.pushBack(currTranslation);
        rotations.pushBack(currRotation);
        frameTime.pushBack(currTime);
    }
    return;
};

void AudioObject::LoadAccs_(const std::string& path)
{
    std::ifstream fin(path);
    float currAcc = 0;
    while(true)
    {
        fin >> currAcc;
        if(!fin.good())
        {
            assert(fin.eof());
            break;
        }
        accelerations.pushBack(currAcc);
    }
    return;
}

}