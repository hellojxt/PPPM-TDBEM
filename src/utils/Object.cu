#include "hip/hip_runtime.h"
#include "Object.h"
#include "hip/hip_vector_types.h"
#include <fstream>
namespace pppm
{
__device__ inline float3 rotate(const float4 q, const float3 v)
{
    float3 u = make_float3(q.x, q.y, q.z);
    float s = q.w;
    return 2.0f * dot(u, v) * u + (s * s - dot(u, u)) * v + 2.0f * s * cross(u, v);
}

__global__ void Transform(GArr<float3> vertices, GArr<float3> standard_vertices, float3 translation, float4 rotation)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= vertices.size())
        return;
    vertices[id] = rotate(rotation, standard_vertices[id]) + translation;
    return;
}

__global__ void FillIf(float *arr, int *judge, float num, size_t size)
{
    int id = (threadIdx.x + blockIdx.x * blockDim.x) * 64;
    for (int i = 0; i < 64 && id + i < size; i++)
    {
        if (judge[id + i] > 0)
        {
            arr[id + i] = num;
        }

        else
            arr[id + i] = 0;
    }
    return;
}

__global__ void FindNearestVertex(GArr<float3> origin_vertices,
                                  GArr<int3> origin_surfaces,
                                  GArr<float3> vertices,
                                  GArr<int3> surfaces,
                                  GArr<int> judge,
                                  GArr<int> selectedVertices)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= selectedVertices.size())
        return;

    int min_dist_id = -1;
    float min_dist = MAX_FLOAT;
    int v_id = selectedVertices[id];
    float3 curr_vert = origin_vertices[v_id];

    for (int f_id = 0; f_id < surfaces.size(); f_id++)
    {
        int3 face = surfaces[f_id];
        float3 verts[3] = {vertices[face.x], vertices[face.y], vertices[face.z]};
        float3 nearest_p = get_nearest_triangle_point(curr_vert, verts[0], verts[1], verts[2]);
        float dist = length(nearest_p - curr_vert);
        if (dist < min_dist)
        {
            min_dist = dist;
            min_dist_id = f_id;
        }
    }
    // printf("cv id:%d, current vertex: (%f, %f, %f), selected face id: %d\n", selectedVertices[id], curr_vert.x,
    //        curr_vert.y, curr_vert.z, min_dist_id);
    atomicAdd(&judge[min_dist_id], 1);
    return;
}

std::pair<CArr<int3>, CArr<float3>> FindAllSurfaces(CArr<int4> &tetrahedrons, CArr<float3> &tetraVerts)
{
    CArr<int3> surfaceTriangles;
    CArr<float3> surfaceNorms;

    struct iVec3Hash
    {
            int operator()(const int3 &vec) const { return (vec.x << 20) + (vec.y << 10) + vec.z; }
    };
    struct iVec3Eq
    {
            bool operator()(const int3 &vec1, const int3 &vec2) const
            {
                return vec1.x == vec2.x && vec1.y == vec2.y && vec1.z == vec2.z;
            }
    };
    struct TriInfo
    {
            int cnt = 0;
            int tetID = 0;
            int exceptVertID = 0;
    };

    std::unordered_map<int3, TriInfo, iVec3Hash, iVec3Eq> candidateTriangles;
    int3 currTri;
    int int3::*int3Members[] = {&int3::x, &int3::y, &int3::z};
    int int4::*int4Members[] = {&int4::x, &int4::y, &int4::z, &int4::w};

    for (int i = 0, size = tetrahedrons.size(); i < size; i++)
    {
        int4 &currVertIDs = tetrahedrons[i];
        for (int i0 = 0; i0 < 4; i0++)
        {
            for (int j = 0, k = 0; j < 4; j++)
            {
                if (j == i0)
                    continue;
                currTri.*(int3Members[k++]) = currVertIDs.*(int4Members[j]);
            }
            auto &currInfo = candidateTriangles[currTri];
            currInfo.cnt++;
            currInfo.tetID = i;
            currInfo.exceptVertID = i0;
        }
    }

    for (auto &candidateTriangle : candidateTriangles)
    {
        if (candidateTriangle.second.cnt != 1)
            continue;

        int3 currTri = candidateTriangle.first;

        int4 &currTet = tetrahedrons[candidateTriangle.second.tetID];
        int exceptVertID = candidateTriangle.second.exceptVertID;

        float3 center = (tetraVerts[currTri.x] + tetraVerts[currTri.y] + tetraVerts[currTri.z]) / 3;
        float3 exceptVec = tetraVerts[currTet.*(int4Members[exceptVertID])] - center;
        float3 e1 = tetraVerts[currTri.y] - tetraVerts[currTri.x], e2 = tetraVerts[currTri.z] - tetraVerts[currTri.x];
        float3 normVec = normalize(cross(e1, e2));
        if (dot(normVec, exceptVec) > 0)
        {
            std::swap(currTri.y, currTri.z);
            normVec = -normVec;
        }
        surfaceTriangles.pushBack(currTri);
        surfaceNorms.pushBack(normVec);
    }

    return {surfaceTriangles, surfaceNorms};
}

void Object::LoadTetMesh_(const std::string &vertsPath,
                          const std::string &tetPath,
                          GArr<float3> &tetVertices,
                          GArr<int3> &tetSurfaces,
                          GArr<float3> &tetSurfaceNorms)
{
    std::ifstream f_verts(vertsPath);
    CArr<float3> cpuTetVertices;
    float3 vert;
    if (!f_verts.good())
    {
        LOG_ERROR("Fail to load tet mesh file.");
        std::exit(EXIT_FAILURE);
    }
    std::string line;
    while (getline(f_verts, line))
    {
        if (line.empty())
            continue;
        std::istringstream iss(line);
        iss >> vert.x >> vert.y >> vert.z;
        cpuTetVertices.pushBack(vert);
    }
    f_verts.close();

    std::ifstream f_tet(tetPath);
    CArr<int4> tetrahedrons;
    float4 tet;
    if (!f_tet.good())
    {
        LOG_ERROR("Fail to load tet mesh file.");
        std::exit(EXIT_FAILURE);
    }
    while (getline(f_tet, line))
    {
        if (line.empty())
            continue;
        std::istringstream iss(line);
        iss >> tet.x >> tet.y >> tet.z >> tet.w;
        int idxs[4] = {F2I(tet.x), F2I(tet.y), F2I(tet.z), F2I(tet.w)};
        std::sort(idxs, idxs + 4);
        tetrahedrons.pushBack(make_int4(idxs[0], idxs[1], idxs[2], idxs[3]));
    }

    tetVertices.assign(cpuTetVertices);
    auto [surfaceTris, surfaceNorms] = FindAllSurfaces(tetrahedrons, cpuTetVertices);
    tetSurfaces.assign(surfaceTris);
    tetSurfaceNorms.assign(surfaceNorms);
    return;
}

void Object::LoadMotion_(const std::string &path,
                         CArr<float3> &translations,
                         CArr<float4> &rotations,
                         CArr<float> &frameTime)
{
    std::ifstream fin(path);

    float currTime = 0;
    float3 currTranslation;
    float4 currRotation;
    if (!fin.good())
    {
        LOG_ERROR("Fail to load displacement file at " << path << "\n");
        return;
    }
    std::string line;
    while (getline(fin, line))
    {
        if (line.empty())
            continue;
        std::istringstream iss(line);
        iss >> currTime >> currTranslation.x >> currTranslation.y >> currTranslation.z >> currRotation.x >>
            currRotation.y >> currRotation.z >> currRotation.w;
        translations.pushBack(currTranslation);
        rotations.pushBack(currRotation);
        frameTime.pushBack(currTime);
    }
    return;
};

void AudioObject::LoadAccs_(const std::string &path)
{
    std::ifstream fin(path);
    if (!fin.good())
    {
        LOG_ERROR("Fail to load acceleration file at " << path << "\n");
        return;
    }
    float currAcc = 0;
    while (true)
    {
        fin >> currAcc;
        if (!fin.good())
        {
            assert(fin.eof());
            break;
        }
        accelerations.pushBack(currAcc);
    }
    return;
}

void AudioObject::LoadCover_(const std::string &path)
{
    std::ifstream fin(path);
    if (!fin.good())
    {
        LOG_ERROR("Fail to load selected vertices file at " << path << "\n");
        return;
    }
    CArr<int> cpuCoverVertices;
    int currVertex = 0;
    while (true)
    {
        fin >> currVertex;
        cpuCoverVertices.pushBack(currVertex);
        if (!fin.good())
        {
            assert(fin.eof());
            break;
        }
    }
    selectedVertices.assign(cpuCoverVertices);
    return;
}

}  // namespace pppm