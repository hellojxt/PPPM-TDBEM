#include "hip/hip_runtime.h"
#include <vector>
#include "array_writer.h"
#include "bem.h"
#include "gui.h"
#include "macro.h"
#include "objIO.h"
#include "pppm.h"
#include "sound_source.h"
#include "visualize.h"
#include "window.h"
#include <filesystem>
#include <fstream>
#include "ghost_cell.h"
#include "pppm.h"
#include "RigidBody.h"
#include "progressbar.h"

using namespace pppm;

int main()
{
    std::string data_dir = DATASET_DIR + std::string("/bowl");
    std::string OUT_DIR = data_dir + "/pppm";
    CHECK_DIR(OUT_DIR);

    BBox bbox;
    bbox.load_from_txt(data_dir + "/bounding_box.txt");
    LOG("bbox: " << bbox.min << " " << bbox.max)

    float3 grid_center = bbox.center();
    float grid_length = bbox.length();
    int res = 20;
    float grid_size = grid_length / res;
    int boundary_size = 3;
    float3 min_pos = grid_center - grid_length / 2 - grid_size * boundary_size;
    res = res + boundary_size * 2;
    float dt = grid_size / (std::sqrt(3) * AIR_WAVE_SPEED * 1.2);
    int frame_rate = 1.0f / dt;
    dt = 1.0f / frame_rate;
    float max_time = 2.5;

    LOG("grid size: " << grid_size)
    LOG("dt: " << dt)
    LOG("frame rate: " << frame_rate)

    PPPMSolver solver(res, grid_size, dt, min_pos);

    RigidBody rigidbody(data_dir, frame_rate, "polystyrene");
    // for debug
    rigidbody.fix_mesh(grid_size, OUT_DIR);
    rigidbody.export_mesh_with_modes(OUT_DIR);
    // rigidbody.export_mesh_sequence(OUT_DIR + "/mesh");
    // rigidbody.export_signal(OUT_DIR, 2.5);
    return 0;

    rigidbody.fix_mesh(grid_size, OUT_DIR);
    rigidbody.move_to_first_impulse();

    int frame_num = (max_time - rigidbody.current_time) / dt;
    auto IMG_DIR = OUT_DIR + "/img/";
    CHECK_DIR(IMG_DIR)
    int3 check_coord = make_int3(res - boundary_size);
    CArr<float> result(frame_num + 2);
    CArr<float> origin_signal(frame_num + 2);
    result.reset();
    origin_signal.reset();
    result[0] = frame_rate;
    origin_signal[0] = frame_rate;

    progressbar bar(frame_num);
    while (bar.get_progress() <= frame_num)
    {
        if (bar.get_progress() == 0)
        {
            solver.set_mesh(rigidbody.tetVertices, rigidbody.tetSurfaces);
        }
        rigidbody.audio_step();
        // if (bar.get_progress() < 20000)
        // {
        //     bar.update();
        //     continue;
        // }
        if (rigidbody.mesh_is_updated)
        {
            solver.update_mesh(rigidbody.tetVertices);
        }
        for (int j = 0; j < rigidbody.cpuQ.size(); j++)
        {
            origin_signal[bar.get_progress() + 1] += rigidbody.cpuQ[j];
        }
        solver.update_grid_and_face(rigidbody.surfaceAccs);
        result[bar.get_progress() + 1] = solver.pg.fdtd.grids[solver.pg.fdtd.t](to_cpu(check_coord));
        bar.update();
        if (bar.get_progress() <= 31005 && bar.get_progress() >= 31000)
        {
            auto sub_img_dir = IMG_DIR + "grid" + std::to_string(bar.get_progress()) + ".png";
            // CHECK_DIR(sub_img_dir)
            save_grid(solver.pg, sub_img_dir, 100);
        }
        // if (bar.get_progress() == 33000)
        //     break;
    }
    std::cout << "Done" << std::endl;
    write_to_txt(OUT_DIR + "/result.txt", result);
    write_to_txt(OUT_DIR + "/origin.txt", origin_signal);
    rigidbody.clear();
    solver.clear();
}
