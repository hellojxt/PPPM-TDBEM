#include "hip/hip_runtime.h"
#include <vector>
#include "array_writer.h"
#include "bem.h"
#include "gui.h"
#include "macro.h"
#include "objIO.h"
#include "pppm.h"
#include "sound_source.h"
#include "visualize.h"
#include "window.h"
#include <filesystem>
#include <fstream>
#include "ghost_cell.h"
#include "pppm.h"
#include "RigidBody.h"
#include "progressbar.h"

using namespace pppm;

int main()
{
    std::string data_dir = DATASET_DIR + std::string("/bowl");
    std::string OUT_DIR = data_dir + "/output/pppm";
    CHECK_DIR(OUT_DIR);
    RigidBody rigidbody(data_dir, "polystyrene");
    BBox bbox = rigidbody.get_bbox();

    float3 grid_center = bbox.center();
    float grid_length = bbox.length() * 2;
    int res = 40;
    float grid_size = grid_length / res;
    float3 min_pos = grid_center - grid_length / 2;
    int frame_rate = 1.01f / (grid_size / std::sqrt(3) / AIR_WAVE_SPEED);
    rigidbody.set_sample_rate(frame_rate);
    float dt = 1.0f / frame_rate;
    float max_time = 2.5;
    LOG("min pos: " << min_pos);
    LOG("grid size: " << grid_size)
    LOG("dt: " << dt)
    LOG("frame rate: " << frame_rate)

    rigidbody.fix_mesh(grid_size, OUT_DIR);
    rigidbody.move_to_first_impulse();

    PPPMSolver solver(res, grid_size, dt, min_pos);

    int frame_num = (max_time - rigidbody.current_time) / dt;
    auto IMG_DIR = OUT_DIR + "/img/";
    CHECK_DIR(IMG_DIR)
    int3 check_coord = make_int3(res / 8 * 7);

    int mute_frame_num = int(rigidbody.current_time / dt);
    CArr<float> result(mute_frame_num + frame_num + 2);
    CArr<float> origin_signal(mute_frame_num + frame_num + 2);
    result.reset();
    origin_signal.reset();
    result[0] = frame_rate;
    origin_signal[0] = frame_rate;

    progressbar bar(frame_num);
    while (bar.get_progress() <= frame_num)
    {
        // if (bar.get_progress() < 20000)
        // {
        //     rigidbody.audio_step();
        //     bar.update();
        //     continue;
        // }
        rigidbody.audio_step();
        if (!solver.mesh_set)
        {
            solver.set_mesh(rigidbody.tetVertices, rigidbody.tetSurfaces);
        }
        else if (rigidbody.mesh_is_updated)
        {
            solver.update_mesh(rigidbody.tetVertices);
        }
        for (int j = 0; j < rigidbody.cpuQ.size(); j++)
        {
            origin_signal[mute_frame_num + bar.get_progress() + 1] += rigidbody.cpuQ[j];
        }
        // if (bar.get_progress() > 10)
        // {
        //     rigidbody.surfaceAccs.reset();
        // }
        solver.update_grid_and_face(rigidbody.surfaceAccs);
        result[mute_frame_num + bar.get_progress() + 1] = solver.pg.fdtd.grids[solver.pg.fdtd.t](to_cpu(check_coord));
        // if (bar.get_progress() <= 5000 && bar.get_progress() % 10 == 0)
        // {
        //     auto sub_filename = IMG_DIR + "grid" + std::to_string(bar.get_progress()) + ".png";
        //     auto sub_img_dir = IMG_DIR + "grid" + std::to_string(bar.get_progress()) + "/";
        //     save_grid(solver.pg, sub_filename, 1000);
        //     // save_all_grid(solver.pg, sub_img_dir, 1);
        // }
        // if (bar.get_progress() == 50000)
        //     break;
        if (isnan(result[mute_frame_num + bar.get_progress() + 1]))
        {
            LOG("NAN")
            break;
        }
        bar.update();
    }
    std::cout << "Done" << std::endl;
    write_to_txt(OUT_DIR + "/result.txt", result);
    write_to_txt(OUT_DIR + "/origin.txt", origin_signal);
    rigidbody.clear();
    solver.clear();
}
