#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include "array_writer.h"
#include "bem.h"
#include "case_generator.h"
#include "gui.h"
#include "macro.h"
#include "objIO.h"
#include "particle_grid.h"
#include "pppm.h"
#include "progressbar.h"
#include "sound_source.h"
#include "visualize.h"
#include "window.h"
#include <filesystem>
#include <fstream>
#include "ghost_cell.h"
#include <sys/stat.h>

using namespace pppm;

#define ALL_TIME 0.015
#define SKIP_TIME 0.01
#define CHECK_BBOX_NUM 2
#define CHECK_BBOX_SIZE \
    {                   \
        0.25, 0.35,     \
    }

void SaveGrid(const std::string &dir, ParticleGrid &grid, float max_value = 1.0f, bool saveAll = false)
{
    CHECK_DIR(dir)
    if (saveAll)
    {
        save_all_grid(grid, dir + "/img/", max_value);
    }
    else
    {
        save_grid(grid, dir + "/img.png", max_value, make_float3(0.5, 0, 0));
    }
}

__global__ void update_surf_acc(SineSource sine,
                                MonoPole mp,
                                GArr<float> surface_accs,
                                GArr<Triangle> triangles,
                                float t)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= surface_accs.size())
        return;
    auto &p = triangles[idx];
    surface_accs[idx] = (mp.neumann(p.center, p.normal) * sine(t)).real();
}

__global__ void collect_ffat_map(GArr3D<float> result, GArr3D<float> grid)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x >= result.size.y || y >= result.size.z)
        return;
    int3 e[3] = {make_int3(1, 0, 0), make_int3(0, 1, 0), make_int3(0, 0, 1)};
    int3 center = make_int3(grid.size.x / 2, grid.size.y / 2, grid.size.z / 2);
    int ffat_width = result.size.y / 2;
    int batch_idx = -1;
    for (int i = 0; i < 3; i++)
        for (int sign = -1; sign <= 1; sign += 2)
        {
            batch_idx++;
            int3 p = center + e[i] * sign * ffat_width;
            int j = x - ffat_width;
            int k = y - ffat_width;
            int3 p1 = p + e[(i + 1) % 3] * j + e[(i + 2) % 3] * k;
            result(batch_idx, x, y) += abs(grid(p1));
        }
}

template <typename T>
void points_test(T &solver, Mesh &mesh, SineSource &sine, MonoPole &mp, std::string dirname)
{
    GArr<float> surface_accs;
    surface_accs.resize(mesh.triangles.size());
    int all_step = ALL_TIME / solver.dt();
    CHECK_DIR(dirname);
    float cost_time = 0;
    int SKIP_FRAME = SKIP_TIME / solver.dt();
    APPEND_TIME(cost_time, solver.set_mesh(mesh.vertices, mesh.triangles), SET_MESH)
    LOG("set mesh cost time: " << cost_time)
    GArr3D<float> check_ffat[CHECK_BBOX_NUM];
    float check_ffat_size[CHECK_BBOX_NUM] = CHECK_BBOX_SIZE;
    for (int i = 0; i < CHECK_BBOX_NUM; i++)
    {
        int ffat_width = check_ffat_size[i] * solver.res();
        ffat_width *= 2;
        check_ffat[i].resize(6, ffat_width, ffat_width);
        check_ffat[i].reset();
    }
    LOG(dirname)
    auto triangles = solver.get_triangles();
    progressbar bar(all_step);
    for (int i = 0; i < all_step; i++)
    {
        bar.update();
        cuExecute(surface_accs.size(), update_surf_acc, sine, mp, surface_accs, triangles, solver.dt() * i);
        APPEND_TIME(cost_time, solver.update_step(surface_accs), UPDATE_STEP)
        if (i > SKIP_FRAME)
            for (int j = 0; j < CHECK_BBOX_NUM; j++)
            {
                cuExecute2D(dim2(check_ffat[j].rows, check_ffat[j].cols), collect_ffat_map, check_ffat[j],
                            solver.get_grid());
            }
    }
    LOG("cost time: " << cost_time)
    for (int i = 0; i < CHECK_BBOX_NUM; i++)
    {
        auto data = check_ffat[i].data.cpu();
        write_to_txt(dirname + "/ffat" + std::to_string(i) + ".txt", data);
    }
    write_to_txt(dirname + "/cost_time.txt", cost_time);
    surface_accs.clear();
    for (int i = 0; i < CHECK_BBOX_NUM; i++)
        check_ffat[i].clear();
}

class GroudTruth
{
    public:
        float grid_size;
        float3 min_pos;
        int grid_dim;
        GArr3D<float> grid;
        int t;
        float delta_t;
        SineSource sine;
        MonoPole mp;
        GArr<Triangle> triangles;
        ParticleGrid pg;
        GroudTruth(float3 min_pos_,
                   int res_,
                   float grid_size_,
                   float dt_,
                   SineSource sine_,
                   MonoPole mp_,
                   GArr<Triangle> triangles_)
        {
            min_pos = min_pos_;
            grid_dim = res_;
            grid_size = grid_size_;
            grid.resize(grid_dim, grid_dim, grid_dim);
            t = 0;
            delta_t = dt_;
            sine = sine_;
            mp = mp_;
            triangles = triangles_;
            pg.init(min_pos_, grid_size_, res_, dt_);
        }
        void set_mesh(CArr<float3> vertices, CArr<int3> triangles) { pg.set_mesh(vertices, triangles); };
        CGPU_FUNC inline float dt() { return delta_t; }
        CGPU_FUNC inline int res() { return grid_dim; }
        GArr3D<float> get_grid() { return grid; }
        CGPU_FUNC inline float3 getCenter(int i, int j, int k) const
        {
            return make_float3((i + 0.5f) * grid_size, (j + 0.5f) * grid_size, (k + 0.5f) * grid_size) + min_pos;
        }
        void update_step(GArr<float> surface_accs);
        void clear()
        {
            grid.clear();
            pg.clear();
        }
        GArr<Triangle> get_triangles() { return triangles; }
};

__global__ void groundtruth_update(SineSource sine, MonoPole mp, GroudTruth gt, float t)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int z = threadIdx.z + blockIdx.z * blockDim.z;
    if (x >= gt.res() || y >= gt.res() || z >= gt.res())
        return;
    float3 p = gt.getCenter(x, y, z);
    gt.grid(x, y, z) = (mp.dirichlet(p) * sine(t)).real();
}

void GroudTruth::update_step(GArr<float> surface_accs)
{
    cuExecute3D(dim3(res(), res(), res()), groundtruth_update, sine, mp, *this, t * delta_t);
    t++;
}

int main(int argc, char *argv[])
{
    std::vector<float> grid_size_list = {0.005, 0.01, 0.015, 0.02, 0.025, 0.03};
    auto dir_name = ROOT_DIR + std::string("dataset/static/");
    if (argc > 1)
        dir_name += std::string(argv[1]) + "/";
    else
        dir_name += "plane/";

    auto OUT_DIR = dir_name + "output/";
    CHECK_DIR(OUT_DIR);
    GArr<Triangle> triangles;

    float scale = 5.0;
    float box_size = 0.7;
    for (auto grid_size : grid_size_list)
    {
        auto OUT_SUB_DIR = OUT_DIR + std::to_string(grid_size) + "/";
        CHECK_DIR(OUT_SUB_DIR);
        auto mesh = Mesh::loadOBJ(dir_name + "mesh.obj");
        mesh.stretch_to(box_size / scale);
        mesh.fix_mesh(grid_size, OUT_SUB_DIR);

        float3 min_pos = mesh.get_center() - box_size / 2;
        float dt = grid_size / (std::sqrt(3) * AIR_WAVE_SPEED * 1.01);
        int res = box_size / grid_size;
        LOG("res: " << res << ", dt: " << dt << ", grid_size: " << grid_size << ", box_size: " << box_size)

        auto sine = SineSource(2 * PI * 1000);
        float wave_number = sine.omega / AIR_WAVE_SPEED;
        auto mp = MonoPole(mesh.get_center(), wave_number);

        // PPPM
        PPPMSolver pppm(res, grid_size, dt, min_pos);
        points_test(pppm, mesh, sine, mp, OUT_SUB_DIR + "/pppm/");
        triangles.assign(pppm.pg.triangles);
        SaveGrid(OUT_SUB_DIR + "/pppm/", pppm.pg);
        pppm.clear();

        // First order Ghost cell
        GhostCellSolver solver1(min_pos, grid_size, res, dt);
        solver1.set_condition_number_threshold(0.0f);
        points_test(solver1, mesh, sine, mp, OUT_SUB_DIR + "/ghostcell1/");
        SaveGrid(OUT_SUB_DIR + "/ghostcell1/", solver1.grid);
        solver1.clear();

        // Second order Ghost cell
        GhostCellSolver solver2(min_pos, grid_size, res, dt);
        solver2.set_condition_number_threshold(25.0f);
        points_test(solver2, mesh, sine, mp, OUT_SUB_DIR + "/ghostcell2/");
        SaveGrid(OUT_SUB_DIR + "/ghostcell2/", solver2.grid);
        solver2.clear();

        // Ground truth
        GroudTruth gt(min_pos, res, grid_size, dt, sine, mp, triangles);
        points_test(gt, mesh, sine, mp, OUT_SUB_DIR + "/groundtruth/");
        gt.pg.fdtd.grids[gt.pg.fdtd.t].assign(gt.grid);
        SaveGrid(OUT_SUB_DIR + "/groundtruth/", gt.pg);
        gt.clear();
    }
}
