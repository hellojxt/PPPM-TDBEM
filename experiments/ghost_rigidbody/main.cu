#include "hip/hip_runtime.h"
#include <vector>
#include "array_writer.h"
#include "bem.h"
#include "gui.h"
#include "macro.h"
#include "objIO.h"
#include "pppm.h"
#include "sound_source.h"
#include "visualize.h"
#include "window.h"
#include <filesystem>
#include <fstream>
#include "ghost_cell.h"
#include "pppm.h"
#include "RigidBody.h"
#include "progressbar.h"

using namespace pppm;

int main()
{
    std::string data_dir = DATASET_DIR + std::string("/bowl");
    std::string OUT_DIR = data_dir + "/output/ghost";
    CHECK_DIR(OUT_DIR);
    RigidBody rigidbody(data_dir, "polystyrene");
    // for debug
    // rigidbody.fix_mesh(grid_size * 2, OUT_DIR);
    // rigidbody.export_mesh_with_modes(OUT_DIR);
    // rigidbody.export_mesh_sequence(OUT_DIR + "/mesh"); // can not used together with export_signal
    // rigidbody.export_signal(OUT_DIR, 2.5);
    // return 0;
    BBox bbox = rigidbody.get_bbox();
    LOG(bbox)

    float3 grid_center = bbox.center();
    float grid_length = bbox.length();
    int res = 40;
    float grid_size = grid_length / res;
    int boundary_size = 3;
    float3 min_pos = grid_center - grid_length / 2 - grid_size * boundary_size;
    res = res + boundary_size * 2;
    float dt = grid_size / (std::sqrt(3) * AIR_WAVE_SPEED * 1.2);
    int frame_rate = 1.0f / dt;
    dt = 1.0f / frame_rate;
    float max_time = 2.5;
    LOG("grid size: " << grid_size)
    LOG("dt: " << dt)
    LOG("frame rate: " << frame_rate)

    rigidbody.set_sample_rate(frame_rate);
    rigidbody.fix_mesh(grid_size * 2, OUT_DIR);
    rigidbody.move_to_first_impulse();
    GhostCellSolver ghost_cell_solver(min_pos, grid_size, res, dt);
    ghost_cell_solver.set_condition_number_threshold(0);

    int frame_num = (max_time - rigidbody.current_time) / dt;
    auto IMG_DIR = OUT_DIR + "/img/";
    CHECK_DIR(IMG_DIR)
    int3 check_coord = make_int3(res - boundary_size);

    int mute_frame_num = int(rigidbody.current_time / dt);
    CArr<float> result(mute_frame_num + frame_num + 2);
    CArr<float> origin_signal(mute_frame_num + frame_num + 2);
    result.reset();
    origin_signal.reset();
    result[0] = frame_rate;
    origin_signal[0] = frame_rate;

    progressbar bar(frame_num);
    while (bar.get_progress() <= frame_num)
    {
        if (bar.get_progress() == 0)
        {
            ghost_cell_solver.set_mesh(rigidbody.tetVertices, rigidbody.tetSurfaces);
        }
        rigidbody.audio_step();
        // if (bar.get_progress() < 70000)
        // {
        //     bar.update();
        //     continue;
        // }
        if (rigidbody.mesh_is_updated)
        {
            ghost_cell_solver.update_mesh(rigidbody.tetVertices);
        }
        for (int j = 0; j < rigidbody.cpuQ.size(); j++)
        {
            origin_signal[mute_frame_num + bar.get_progress() + 1] += rigidbody.cpuQ[j];
        }
        if (bar.get_progress() > 10)
        {
            rigidbody.surfaceAccs.reset();
        }
        ghost_cell_solver.update(rigidbody.surfaceAccs);
        result[mute_frame_num + bar.get_progress() + 1] =
            ghost_cell_solver.grid.fdtd.grids[ghost_cell_solver.grid.fdtd.t](to_cpu(check_coord));
        // LOG("result: " << result[mute_frame_num + bar.get_progress() + 1])
        bar.update();
        // if (mute_frame_num + bar.get_progress() <= 90000 && mute_frame_num + bar.get_progress() >= 78000 &&
        //     (mute_frame_num + bar.get_progress()) % 100 == 0)
        // {
        //     auto sub_filename = IMG_DIR + "grid" + std::to_string(mute_frame_num + bar.get_progress()) + ".png";
        //     auto sub_img_dir = IMG_DIR + "grid" + std::to_string(mute_frame_num + bar.get_progress()) + "/";
        //     save_grid(ghost_cell_solver.grid, sub_filename, 200);
        //     // CHECK_DIR(sub_img_dir)
        //     // save_all_grid(ghost_cell_solver.grid, sub_img_dir, 100);
        // }
        // if (mute_frame_num + bar.get_progress() == 80000)
        //     break;
    }
    std::cout << "Done" << std::endl;
    write_to_txt(OUT_DIR + "/result.txt", result);
    write_to_txt(OUT_DIR + "/origin.txt", origin_signal);
    rigidbody.clear();
    ghost_cell_solver.clear();
}

// void test_Ghost()
// {
//     std::string OUT_DIR = EXP_DIR + std::string("rigidbody/output/") + obj_name + "/ghost";
//     float3 grid_center = make_float3(0.0, 0.08, 0.015);
//     float grid_length = 0.005 * 50;
//     int res = 50;
//     float grid_size = grid_length / res;
//     int boundary_size = 5;
//     float3 min_pos = grid_center - grid_length / 2 - grid_size * boundary_size;
//     res = res + boundary_size * 2;
//     float dt = grid_size / (std::sqrt(3) * AIR_WAVE_SPEED * 1.2);
//     int frame_rate = 1.0f / dt;
//     dt = 1.0f / frame_rate;
//     float max_time = 2.5;

//     printf("grid size: %f\n", grid_size);
//     printf("dt: %e\n", dt);
//     printf("frame rate: %d\n", frame_rate);

//     GhostCellSolver ghost_cell_solver(min_pos, grid_size, res, dt);
//     ghost_cell_solver.set_condition_number_threshold(15);
//     float3 reflect_center = make_float3(0, -grid_size / 2, 0);
//     int3 reflect_coord = ghost_cell_solver.grid.getGridCoord(reflect_center);
//     int3 reflect_normal = make_int3(0, 1, 0);
//     // ghost_cell_solver.grid.fdtd.set_reflect_boundary(reflect_coord, reflect_normal);

//     RigidBody rigidbody(DATASET_DIR + obj_name, frame_rate, "polystyrene");
//     rigidbody.fix_mesh(2e-2, OUT_DIR);
//     rigidbody.move_to_first_impulse();
//     int frame_num = (max_time - rigidbody.current_time) / dt;
//     auto IMG_DIR = OUT_DIR + "/img/";
//     CHECK_DIR(IMG_DIR)
//     int3 check_coord = make_int3(res - boundary_size);
//     CArr<float> resultGhost(frame_num + 2);
//     CArr<float> origin_signal(frame_num + 2);
//     resultGhost.reset();
//     origin_signal.reset();
//     resultGhost[0] = frame_rate;
//     origin_signal[0] = frame_rate;

//     progressbar bar(frame_num);
//     while (bar.get_progress() <= frame_num)
//     {
//         if (bar.get_progress() == 0)
//         {
//             ghost_cell_solver.set_mesh(rigidbody.tetVertices, rigidbody.tetSurfaces);
//         }
//         rigidbody.audio_step();
//         if (rigidbody.mesh_is_updated)
//         {
//             ghost_cell_solver.update_mesh(rigidbody.tetVertices);
//         }
//         for (int j = 0; j < rigidbody.cpuQ.size(); j++)
//         {
//             origin_signal[bar.get_progress() + 1] += rigidbody.cpuQ[j];
//         }
//         // if (bar.get_progress() > 0)
//         //     rigidbody.surfaceAccs.reset();
//         ghost_cell_solver.update(rigidbody.surfaceAccs);
//         resultGhost[bar.get_progress() + 1] =
//             ghost_cell_solver.grid.fdtd.grids[ghost_cell_solver.grid.fdtd.t](to_cpu(check_coord));
//         bar.update();
//         // if (bar.get_progress() < 200)
//         // {
//         //     save_grid(ghost_cell_solver.grid, IMG_DIR + "grid" + std::to_string(bar.get_progress()) + ".png",
//         1e-4f);
//         // }
//         // else
//         //     break;
//         // if (bar.get_progress() > 5000)
//         //     break;
//     }
//     std::cout << "Done" << std::endl;
//     write_to_txt(OUT_DIR + "/result.txt", resultGhost);
//     write_to_txt(OUT_DIR + "/origin.txt", origin_signal);
//     rigidbody.clear();
//     ghost_cell_solver.clear();
// }

// int main()
// {
//     CHECK_DIR(EXP_DIR + std::string("rigidbody/output/") + obj_name);
//     test_PPPM();
//     // test_Ghost();
//     // RigidBody rigidbody(DATASET_DIR + obj_name, 44100, "polystyrene");
//     // rigidbody.export_signal(EXP_DIR + std::string("rigidbody/output/") + obj_name + "/ghost", 2.5);
//     // rigidbody.export_mesh_with_modes(EXP_DIR + std::string("rigidbody/output/") + obj_name + "/ghost");
//     return 0;
// }
