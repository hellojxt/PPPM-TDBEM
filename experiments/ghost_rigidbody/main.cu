#include "hip/hip_runtime.h"
#include <vector>
#include "array_writer.h"
#include "bem.h"
#include "gui.h"
#include "macro.h"
#include "objIO.h"
#include "pppm.h"
#include "sound_source.h"
#include "visualize.h"
#include "window.h"
#include <filesystem>
#include <fstream>
#include "ghost_cell.h"
#include "pppm.h"
#include "RigidBody.h"
#include "progressbar.h"

using namespace pppm;

int main()
{
    std::string data_dir = DATASET_DIR + std::string("/bowl");
    std::string OUT_DIR = data_dir + "/output/ghost";
    CHECK_DIR(OUT_DIR);
    RigidBody rigidbody(data_dir, "polystyrene");
    BBox bbox = rigidbody.get_bbox();
    LOG(bbox)
    float3 grid_center = bbox.center();
    float grid_length = bbox.length() * 2;
    int res = 128;
    float grid_size = grid_length / res;
    float3 min_pos = grid_center - grid_length / 2;
    int frame_rate = 1.01f / (grid_size / std::sqrt(3) / AIR_WAVE_SPEED);
    rigidbody.set_sample_rate(frame_rate);
    float dt = 1.0f / frame_rate;
    float max_time = 2.5;
    LOG("min pos: " << min_pos);
    LOG("grid size: " << grid_size)
    LOG("dt: " << dt)
    LOG("frame rate: " << frame_rate)
    rigidbody.move_to_first_impulse();
    GhostCellSolver ghost_cell_solver(min_pos, grid_size, res, dt);
    ghost_cell_solver.set_condition_number_threshold(0);

    int frame_num = (max_time - rigidbody.current_time) / dt;
    auto IMG_DIR = OUT_DIR + "/img/";
    CHECK_DIR(IMG_DIR)
    int3 check_coord = make_int3(res / 8 * 7);

    int mute_frame_num = int(rigidbody.current_time / dt);
    CArr<float> result(mute_frame_num + frame_num + 2);
    CArr<float> origin_signal(mute_frame_num + frame_num + 2);
    result.reset();
    origin_signal.reset();
    result[0] = frame_rate;
    origin_signal[0] = frame_rate;

    progressbar bar(frame_num);

    while (bar.get_progress() <= frame_num)
    {
        // printf("frame: %d\n", bar.get_progress());
        // if (mute_frame_num + bar.get_progress() < 141220)
        // {
        //     bar.update();
        //     rigidbody.audio_step();
        //     continue;
        // }
        rigidbody.audio_step();
        if (!ghost_cell_solver.mesh_set)
        {
            ghost_cell_solver.set_mesh(rigidbody.tetVertices, rigidbody.tetSurfaces);
        }
        else if (rigidbody.mesh_is_updated)
        {
            ghost_cell_solver.update_mesh(rigidbody.tetVertices);
        }
        for (int j = 0; j < rigidbody.cpuQ.size(); j++)
        {
            origin_signal[mute_frame_num + bar.get_progress() + 1] += rigidbody.cpuQ[j];
        }
        // if (bar.get_progress() > 0)
        // {
        //     // rigidbody.surfaceAccs.reset();
        //     ghost_cell_solver.grid.fdtd.step();
        // }
        //
        if (ghost_cell_solver.fresh_error.cpu()[0] == 1)
        {
            auto sub_filename = IMG_DIR + "grid" + std::to_string(mute_frame_num + bar.get_progress()) + ".png";
            auto sub_img_dir = IMG_DIR + "grid" + std::to_string(mute_frame_num + bar.get_progress()) + "/";
            // save_grid(ghost_cell_solver.grid, sub_filename, 10000.0f);
            save_all_grid(ghost_cell_solver.grid, sub_img_dir, 1e10);
            return 0;
        }
        ghost_cell_solver.update(rigidbody.surfaceAccs);
        result[mute_frame_num + bar.get_progress() + 1] =
            ghost_cell_solver.grid.fdtd.grids[ghost_cell_solver.grid.fdtd.t](to_cpu(check_coord));

        // if (bar.get_progress() <= 20000 && bar.get_progress() % 10 == 0)
        // {
        //     auto sub_filename = IMG_DIR + "grid" + std::to_string(ghost_cell_solver.grid.fdtd.t) + ".png";
        //     auto sub_img_dir = IMG_DIR + "grid" + std::to_string(ghost_cell_solver.grid.fdtd.t) + "/";
        //     auto sub_cell_data_dir = IMG_DIR + "grid" + std::to_string(ghost_cell_solver.grid.fdtd.t) + "_cell/";
        //     save_grid(ghost_cell_solver.grid, sub_filename, 1000.0f);
        //     // save_all_grid(ghost_cell_solver.grid, sub_img_dir, 1e10);
        //     // save_cell_data(ghost_cell_solver, sub_cell_data_dir);
        // }
        // if (bar.get_progress() == 100000)
        // {
        //     break;
        // }

        if (isnan(result[mute_frame_num + bar.get_progress() + 1]))
        {
            LOG("NAN")
            break;
        }
        bar.update();
    }
    std::cout << "Done" << std::endl;
    write_to_txt(OUT_DIR + "/result.txt", result);
    write_to_txt(OUT_DIR + "/origin.txt", origin_signal);
    rigidbody.clear();
    ghost_cell_solver.clear();
}
